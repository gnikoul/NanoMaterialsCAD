#include "hip/hip_runtime.h"
#define JGN_COMMANDS_CPP


#include "stdafx.h"
#include "JGN_Windows.h"
#include "JGN_Commands.h"

#include <omp.h>
#include <hip/hip_runtime.h>
#include <>


void jgnCommands(LPTSTR ttt, int d)
{
	
	FILE* periodic_table;
	int isngtv = 0;
	//char *test1 = "Vector(";
	for (i = 0; i < 7; i++)
	{
		if (test1[0][i] == ttt[i])
		{

		}
		else
		{
			i = 100;

		}
	}
	if (i == 7)
	{

		okrender = 1;
		CustomSurfacesCount++;

		CustomSurfaces = (float**)realloc(CustomSurfaces, sizeof(float*)*CustomSurfacesCount);
		CustomSurfaces[CustomSurfacesCount - 1] = (float*)malloc(sizeof(float) * 4);
		CustomSurfaces[CustomSurfacesCount - 1][0] = 0;
		CustomSurfaces[CustomSurfacesCount - 1][1] = 0;
		CustomSurfaces[CustomSurfacesCount - 1][2] = 0;
		CustomSurfaces[CustomSurfacesCount - 1][3] = 0;
		help = (char*)(ttt + 7);
		if (help[0] >= 48 && help[0] <= 57)
		{
			crystalh = help[0] - 48;
		}
		else if (help[0] == '-')
		{
			isngtv = 1;
		}
		else
		{
			okrender = 0;
			return;
		}
		help = help + 2;
		loop = 0;
		while (help[0] != ',' && loop < 6)
		{
			loop++;
			if (help[0] >= 48 && help[0] <= 57)
			{
				crystalh = 10 * crystalh + help[0] - 48;
				help += 2;
			}
			else
			{
				okrender = 0;
				return;

			}
		}
		loop = 0;
		if (isngtv)
		{
			crystalh = -crystalh;
		}

		CustomSurfaces[CustomSurfacesCount - 1][0] += ijk[0][0] * crystalh;
		CustomSurfaces[CustomSurfacesCount - 1][1] += ijk[0][1] * crystalh;
		CustomSurfaces[CustomSurfacesCount - 1][2] += ijk[0][2] * crystalh;

		help += 2;
		isngtv = 0;


		if (help[0] >= 48 && help[0] <= 57)
		{
			crystalk = help[0] - 48;
		}
		else if (help[0] == '-')
		{
			isngtv = 1;
		}
		else
		{
			okrender = 0;
			return;
		}
		help = help + 2;
		while (help[0] != ',' && loop < 6)

		{
			loop++;
			if (help[0] >= 48 && help[0] <= 57)
			{
				crystalk = 10 * crystalk + help[0] - 48;
				help += 2;
			}
			else
			{
				okrender = 0;
				return;

			}
		}
		loop = 0;
		if (isngtv)
		{
			crystalk = -crystalk;
		}
		help += 2;
		isngtv = 0;

		CustomSurfaces[CustomSurfacesCount - 1][0] += ijk[1][0] * crystalk;
		CustomSurfaces[CustomSurfacesCount - 1][1] += ijk[1][1] * crystalk;
		CustomSurfaces[CustomSurfacesCount - 1][2] += ijk[1][2] * crystalk;

		if (help[0] >= 48 && help[0] <= 57)
		{
			crystall = help[0] - 48;
		}
		else if (help[0] == '-')
		{
			isngtv = 1;
		}
		else
		{
			okrender = 0;
			return;
		}
		help = help + 2;
		loop = 0;
		while (help[0] != ',' && loop < 6)

		{
			loop++;
			if (help[0] >= 48 && help[0] <= 57)
			{
				crystall = 10 * crystall + help[0] - 48;
				help += 2;
			}
			else
			{
				okrender = 0;
				return;
			}
		}
		loop = 0;
		if (isngtv)
		{
			crystall = -crystall;
		}

		CustomSurfaces[CustomSurfacesCount - 1][0] += ijk[2][0] * crystall;
		CustomSurfaces[CustomSurfacesCount - 1][1] += ijk[2][1] * crystall;
		CustomSurfaces[CustomSurfacesCount - 1][2] += ijk[2][2] * crystall;

		helping1 = CustomSurfaces[CustomSurfacesCount - 1][0];
		helping2 = CustomSurfaces[CustomSurfacesCount - 1][1];
		helping3 = CustomSurfaces[CustomSurfacesCount - 1][2];

		CustomSurfaces[CustomSurfacesCount - 1][0] = CustomSurfaces[CustomSurfacesCount - 1][0] / sqrt(pow(helping1, 2) + pow(helping2, 2) + pow(helping3, 2));
		CustomSurfaces[CustomSurfacesCount - 1][1] = CustomSurfaces[CustomSurfacesCount - 1][1] / sqrt(pow(helping1, 2) + pow(helping2, 2) + pow(helping3, 2));
		CustomSurfaces[CustomSurfacesCount - 1][2] = CustomSurfaces[CustomSurfacesCount - 1][2] / sqrt(pow(helping1, 2) + pow(helping2, 2) + pow(helping3, 2));

		help += 2;
		isngtv = 0;



		if (help[0] >= 48 && help[0] <= 57)
		{
			CustomSurfaces[CustomSurfacesCount - 1][3] = help[0] - 48;
		}
		else if (help[0] == '-')
		{
			isngtv = 1;
		}
		else
		{
			okrender = 0;
			return;

		}
		help = help + 2;
		while (help[0] != ')' && loop < 6)

		{
			loop++;
			if (help[0] >= 48 && help[0] <= 57)
			{
				CustomSurfaces[CustomSurfacesCount - 1][3] = 10 * CustomSurfaces[CustomSurfacesCount - 1][3] + help[0] - 48;
				help += 2;
			}
			else
			{
				okrender = 0;
				return;
			}
		}
		loop = 0;
		if (isngtv)
		{
			CustomSurfaces[CustomSurfacesCount - 1][3] = -CustomSurfaces[CustomSurfacesCount - 1][3];
		}
		help += 2;
		isngtv = 0;


		if (okrender == 0)
		{
			CustomSurfacesCount--;
		}

		goto peintit;


	}
	//render on/off
	for (i = 0; i < 6; i++)
	{
		if (test1[9][i] == ttt[i])
		{

		}
		else
		{
			i = 100;
		}
	}
	if (i == 6)
	{
		okrender = 1;
		help = (char*)(ttt + 7);

		jgn::string option = LPTSTR2string((LPTSTR)help, ')', 2);

		if (option == "on")
		{
			render_is_on = true;
		}
		else if (option == "of")
		{
			render_is_on = false;
		}
		else
		{
			okrender = 0;
			return;
		}
		goto peintit;

	}
	//"rand("
	for (i = 0; i < 5; i++)
	{
		if (test1[8][i] == ttt[i])
		{

		}
		else
		{
			i = 100;
		}
	}
	if (i == 5)
	{
		okrender = 1;

		float r = 0;

		help = (char*)(ttt + 5);

		jgn::string rstr = LPTSTR2string((LPTSTR)help, ')');

		if (rstr.isnumber())
		{
			ole = t * sized[0] * sized[1] * sized[2];

			r = stof(rstr);
			srand(time(NULL));
#pragma omp parallel for firstprivate(ole, r)
			for (int ole1 = 0; ole1 < ole; ole1++)
			{
				crystal[2 + 5 * ole1] += r * (2 * (rand() / (float)RAND_MAX) - 1);
				crystal[3 + 5 * ole1] += r * (2 * (rand() / (float)RAND_MAX) - 1);
				crystal[4 + 5 * ole1] += r * (2 * (rand() / (float)RAND_MAX) - 1);
			}
		}
		else
		{
			okrender = 0;
			return;
		}

		goto peintit;

	}
	//"Plane("
	for (i = 0; i < 6; i++)
	{
		if (test1[4][i] == ttt[i])
		{

		}
		else
		{
			i = 100;
		}
	}
	if (i == 6)
	{

		okrender = 1;
		CustomSurfacesCount++;

		CustomSurfaces = (float**)realloc(CustomSurfaces, sizeof(float*)*CustomSurfacesCount);
		CustomSurfaces[CustomSurfacesCount - 1] = (float*)malloc(sizeof(float) * 4);
		CustomSurfaces[CustomSurfacesCount - 1][0] = 0;
		CustomSurfaces[CustomSurfacesCount - 1][1] = 0;
		CustomSurfaces[CustomSurfacesCount - 1][2] = 0;
		CustomSurfaces[CustomSurfacesCount - 1][3] = 0;
		crystalh = 0;
		crystalk = 0;
		crystall = 0;
		help = (char*)(ttt + 6);
		if (help[0] >= 48 && help[0] <= 57)
		{
			crystalh = help[0] - 48;
		}
		else if (help[0] == '-')
		{
			isngtv = 1;
		}
		else
		{
			okrender = 0;
			return;
		}
		help = help + 2;
		loop = 0;
		while (help[0] != ',' && loop < 6)
		{
			loop++;
			if (help[0] >= 48 && help[0] <= 57)
			{
				crystalh = 10 * crystalh + help[0] - 48;
				help += 2;
			}
			else
			{
				okrender = 0;
				return;


			}
		}
		loop = 0;
		if (isngtv)
		{
			crystalh = -crystalh;
		}




		help += 2;
		isngtv = 0;


		if (help[0] >= 48 && help[0] <= 57)
		{
			crystalk = help[0] - 48;
		}
		else if (help[0] == '-')
		{
			isngtv = 1;
		}
		else
		{
			okrender = 0;
			return;

		}
		help = help + 2;
		while (help[0] != ',' && loop < 6)

		{
			loop++;
			if (help[0] >= 48 && help[0] <= 57)
			{
				crystalk = 10 * crystalk + help[0] - 48;
				help += 2;
			}
			else
			{
				okrender = 0;
				return;

			}
		}
		loop = 0;
		if (isngtv)
		{
			crystalk = -crystalk;
		}
		help += 2;
		isngtv = 0;





		if (help[0] >= 48 && help[0] <= 57)
		{
			crystall = help[0] - 48;
		}
		else if (help[0] == '-')
		{
			isngtv = 1;
		}
		else
		{
			okrender = 0;
			return;
		}
		help = help + 2;
		loop = 0;
		while (help[0] != ',' && loop < 6)

		{
			loop++;
			if (help[0] >= 48 && help[0] <= 57)
			{
				crystall = 10 * crystall + help[0] - 48;
				help += 2;
			}
			else
			{
				okrender = 0;
				return;

			}
		}
		loop = 0;
		if (isngtv)
		{
			crystall = -crystall;
		}

		if (crystalh == 0)
		{
			if (crystalk == 0)
			{
				if (crystall == 0)//000 ok
				{
					CustomSurfacesCount--;
					okrender = 0;
					return;
				}
				else//00l ok
				{
					CustomSurfaces[CustomSurfacesCount - 1][0] = ijk[0][1] * ijk[1][2] - ijk[0][2] * ijk[1][1];
					CustomSurfaces[CustomSurfacesCount - 1][1] = -ijk[0][0] * ijk[1][2] + ijk[0][2] * ijk[1][0];
					CustomSurfaces[CustomSurfacesCount - 1][2] = ijk[0][0] * ijk[1][1] - ijk[0][1] * ijk[1][0];


					CustomSurfaces[CustomSurfacesCount - 1][0] = crystall * CustomSurfaces[CustomSurfacesCount - 1][0];
					CustomSurfaces[CustomSurfacesCount - 1][1] = crystall * CustomSurfaces[CustomSurfacesCount - 1][1];
					CustomSurfaces[CustomSurfacesCount - 1][2] = crystall * CustomSurfaces[CustomSurfacesCount - 1][2];

				}
			}
			else
			{
				if (crystall == 0)//0k0 ok
				{
					CustomSurfaces[CustomSurfacesCount - 1][0] = -ijk[0][1] * ijk[2][2] + ijk[0][2] * ijk[2][1];
					CustomSurfaces[CustomSurfacesCount - 1][1] = ijk[0][0] * ijk[2][2] - ijk[0][2] * ijk[2][0];
					CustomSurfaces[CustomSurfacesCount - 1][2] = -ijk[0][0] * ijk[2][1] + ijk[0][1] * ijk[2][0];

					CustomSurfaces[CustomSurfacesCount - 1][0] = crystalk * CustomSurfaces[CustomSurfacesCount - 1][0];
					CustomSurfaces[CustomSurfacesCount - 1][1] = crystalk * CustomSurfaces[CustomSurfacesCount - 1][1];
					CustomSurfaces[CustomSurfacesCount - 1][2] = crystalk * CustomSurfaces[CustomSurfacesCount - 1][2];

				}
				else//0kl ok
				{
					CustomSurfaces[CustomSurfacesCount - 1][0] = ijk[0][1] * (ijk[1][2] / crystalk - ijk[2][2] / crystall) - ijk[0][2] * (ijk[1][1] / crystalk - ijk[2][1] / crystall);
					CustomSurfaces[CustomSurfacesCount - 1][1] = -ijk[0][0] * (ijk[1][2] / crystalk - ijk[2][2] / crystall) + ijk[0][2] * (ijk[1][0] / crystalk - ijk[2][0] / crystall);
					CustomSurfaces[CustomSurfacesCount - 1][2] = ijk[0][0] * (ijk[1][1] / crystalk - ijk[2][1] / crystall) - ijk[0][1] * (ijk[1][0] / crystalk - ijk[2][0] / crystall);

					CustomSurfaces[CustomSurfacesCount - 1][0] = crystalk * crystall*CustomSurfaces[CustomSurfacesCount - 1][0];
					CustomSurfaces[CustomSurfacesCount - 1][1] = crystalk * crystall*CustomSurfaces[CustomSurfacesCount - 1][1];
					CustomSurfaces[CustomSurfacesCount - 1][2] = crystalk * crystall*CustomSurfaces[CustomSurfacesCount - 1][2];
				}
			}
		}
		else
		{
			if (crystalk == 0)
			{
				if (crystall == 0)//h00 ok
				{
					CustomSurfaces[CustomSurfacesCount - 1][0] = ijk[1][1] * ijk[2][2] - ijk[1][2] * ijk[2][1];
					CustomSurfaces[CustomSurfacesCount - 1][1] = -ijk[1][0] * ijk[2][2] + ijk[1][2] * ijk[2][0];
					CustomSurfaces[CustomSurfacesCount - 1][2] = ijk[1][0] * ijk[2][1] - ijk[1][1] * ijk[2][0];

					CustomSurfaces[CustomSurfacesCount - 1][0] = crystalh * CustomSurfaces[CustomSurfacesCount - 1][0];
					CustomSurfaces[CustomSurfacesCount - 1][1] = crystalh * CustomSurfaces[CustomSurfacesCount - 1][1];
					CustomSurfaces[CustomSurfacesCount - 1][2] = crystalh * CustomSurfaces[CustomSurfacesCount - 1][2];

				}
				else//h0l ok
				{


					CustomSurfaces[CustomSurfacesCount - 1][0] = ijk[1][1] * (ijk[0][2] / crystalh - ijk[2][2] / crystall) - ijk[1][2] * (ijk[0][1] / crystalh - ijk[2][1] / crystall);
					CustomSurfaces[CustomSurfacesCount - 1][1] = -ijk[1][0] * (ijk[0][2] / crystalh - ijk[2][2] / crystall) + ijk[1][2] * (ijk[0][0] / crystalh - ijk[2][0] / crystall);
					CustomSurfaces[CustomSurfacesCount - 1][2] = ijk[1][0] * (ijk[0][1] / crystalh - ijk[2][1] / crystall) - ijk[1][1] * (ijk[0][0] / crystalh - ijk[2][0] / crystall);

					CustomSurfaces[CustomSurfacesCount - 1][0] = -crystalh * crystall*CustomSurfaces[CustomSurfacesCount - 1][0];
					CustomSurfaces[CustomSurfacesCount - 1][1] = -crystalh * crystall*CustomSurfaces[CustomSurfacesCount - 1][1];
					CustomSurfaces[CustomSurfacesCount - 1][2] = -crystalh * crystall*CustomSurfaces[CustomSurfacesCount - 1][2];


				}
			}
			else
			{
				if (crystall == 0)//hk0 ok
				{
					CustomSurfaces[CustomSurfacesCount - 1][0] = ijk[2][1] * (ijk[0][2] / crystalh - ijk[1][2] / crystalk) - ijk[2][2] * (ijk[0][1] / crystalh - ijk[1][1] / crystalk);
					CustomSurfaces[CustomSurfacesCount - 1][1] = -ijk[2][0] * (ijk[0][2] / crystalh - ijk[1][2] / crystalk) + ijk[2][2] * (ijk[0][0] / crystalh - ijk[1][0] / crystalk);
					CustomSurfaces[CustomSurfacesCount - 1][2] = ijk[2][0] * (ijk[0][1] / crystalh - ijk[1][1] / crystalk) - ijk[2][1] * (ijk[0][0] / crystalh - ijk[1][0] / crystalk);

					CustomSurfaces[CustomSurfacesCount - 1][0] = crystalk * crystalh*CustomSurfaces[CustomSurfacesCount - 1][0];
					CustomSurfaces[CustomSurfacesCount - 1][1] = crystalk * crystalh*CustomSurfaces[CustomSurfacesCount - 1][1];
					CustomSurfaces[CustomSurfacesCount - 1][2] = crystalk * crystalh*CustomSurfaces[CustomSurfacesCount - 1][2];
				}
				else//hkl
				{
					CustomSurfaces[CustomSurfacesCount - 1][0] = (ijk[0][1] / crystalh - ijk[2][1] / crystall) * (ijk[0][2] / crystalh - ijk[1][2] / crystalk) - (ijk[0][2] / crystalh - ijk[2][2] / crystall) * (ijk[0][1] / crystalh - ijk[1][1] / crystalk);
					CustomSurfaces[CustomSurfacesCount - 1][1] = -(ijk[0][0] / crystalh - ijk[2][0] / crystall) * (ijk[0][2] / crystalh - ijk[1][2] / crystalk) + (ijk[0][2] / crystalh - ijk[2][2] / crystall) * (ijk[0][0] / crystalh - ijk[1][0] / crystalk);
					CustomSurfaces[CustomSurfacesCount - 1][2] = (ijk[0][0] / crystalh - ijk[2][0] / crystall) * (ijk[0][1] / crystalh - ijk[1][1] / crystalk) - (ijk[0][1] / crystalh - ijk[2][1] / crystall) * (ijk[0][0] / crystalh - ijk[1][0] / crystalk);

					CustomSurfaces[CustomSurfacesCount - 1][0] = -crystall * crystalk*crystalh*CustomSurfaces[CustomSurfacesCount - 1][0];
					CustomSurfaces[CustomSurfacesCount - 1][1] = -crystall * crystalk*crystalh*CustomSurfaces[CustomSurfacesCount - 1][1];
					CustomSurfaces[CustomSurfacesCount - 1][2] = -crystall * crystalk*crystalh*CustomSurfaces[CustomSurfacesCount - 1][2];

				}
			}
		}



		/*if (crystalh == 0)
		{
		if (JGN_Det3x3(ijk[0][0], ijk[0][1], ijk[0][2], ijk[0][2], 0, -ijk[0][0], ijk[0][1], -ijk[0][0], 0) != 0)
		{
		CustomSurfaces[CustomSurfacesCount - 1][0] = JGN_Det3x3(0, ijk[0][1], ijk[0][2], helping1*ijk[0][2] - helping3*ijk[0][0], 0, -ijk[0][0], helping1*ijk[0][1] - helping2*ijk[0][0], -ijk[0][0], 0) / JGN_Det3x3(ijk[0][0], ijk[0][1], ijk[0][2], ijk[0][2], 0, -ijk[0][0], ijk[0][1], -ijk[0][0], 0);
		CustomSurfaces[CustomSurfacesCount - 1][1] = JGN_Det3x3(ijk[0][0], 0, ijk[0][2], ijk[0][2], helping1*ijk[0][2] - helping3*ijk[0][0] , -ijk[0][0], ijk[0][1], helping1*ijk[0][1] - helping2*ijk[0][0], 0) / JGN_Det3x3(ijk[0][0], ijk[0][1], ijk[0][2], ijk[0][2], 0, -ijk[0][0], ijk[0][1], -ijk[0][0], 0);
		CustomSurfaces[CustomSurfacesCount - 1][2] = JGN_Det3x3(ijk[0][0], ijk[0][1], 0, ijk[0][2], 0, helping1*ijk[0][2] - helping3*ijk[0][0] , ijk[0][1], -ijk[0][0], helping1*ijk[0][1] - helping2*ijk[0][0]) / JGN_Det3x3(ijk[0][0], ijk[0][1], ijk[0][2], ijk[0][2], 0, -ijk[0][0], ijk[0][1], -ijk[0][0], 0);
		}
		else if (JGN_Det3x3(0, ijk[0][2], -ijk[0][1], ijk[0][0], ijk[0][1], ijk[0][2], ijk[0][1], -ijk[0][0], 0) != 0)
		{
		CustomSurfaces[CustomSurfacesCount - 1][0] = JGN_Det3x3(helping2*ijk[0][2]-helping3*ijk[0][1], ijk[0][2], -ijk[0][1], 0, ijk[0][1], ijk[0][2], helping1*ijk[0][1]-helping2*ijk[0][0], -ijk[0][0], 0) / JGN_Det3x3(0, ijk[0][2], -ijk[0][1], ijk[0][0], ijk[0][1], ijk[0][2], ijk[0][1], -ijk[0][0], 0);
		CustomSurfaces[CustomSurfacesCount - 1][1] = JGN_Det3x3(0, helping2*ijk[0][2] - helping3*ijk[0][1], -ijk[0][1], ijk[0][0], 0, ijk[0][2], ijk[0][1], helping1*ijk[0][1] - helping2*ijk[0][0] , 0) / JGN_Det3x3(0, ijk[0][2], -ijk[0][1], ijk[0][0], ijk[0][1], ijk[0][2], ijk[0][1], -ijk[0][0], 0);
		CustomSurfaces[CustomSurfacesCount - 1][2] = JGN_Det3x3(0, ijk[0][2], helping2*ijk[0][2] - helping3*ijk[0][1], ijk[0][0], ijk[0][1], 0, ijk[0][1], -ijk[0][0], helping1*ijk[0][1] - helping2*ijk[0][0]) / JGN_Det3x3(0, ijk[0][2], -ijk[0][1], ijk[0][0], ijk[0][1], ijk[0][2], ijk[0][1], -ijk[0][0], 0);
		}
		else if (JGN_Det3x3(0, ijk[0][2], -ijk[0][1], ijk[0][2], 0, -ijk[0][0], ijk[0][0], ijk[0][1], ijk[0][2]) != 0)
		{
		CustomSurfaces[CustomSurfacesCount - 1][0] = JGN_Det3x3(helping2*ijk[0][2] - helping3*ijk[0][1], ijk[0][2], -ijk[0][1], helping1*ijk[0][2] - helping3*ijk[0][0], 0, -ijk[0][0], 0, ijk[0][1], ijk[0][2]) / JGN_Det3x3(0, ijk[0][2], -ijk[0][1], ijk[0][2], 0, -ijk[0][0], ijk[0][0], ijk[0][1], ijk[0][2]);
		CustomSurfaces[CustomSurfacesCount - 1][1] = JGN_Det3x3(0, helping2*ijk[0][2] - helping3*ijk[0][1], -ijk[0][1], ijk[0][2], helping1*ijk[0][2] - helping3*ijk[0][0], -ijk[0][0], ijk[0][0], 0, ijk[0][2]) / JGN_Det3x3(0, ijk[0][2], -ijk[0][1], ijk[0][2], 0, -ijk[0][0], ijk[0][0], ijk[0][1], ijk[0][2]);
		CustomSurfaces[CustomSurfacesCount - 1][2] = JGN_Det3x3(0, ijk[0][2], helping2*ijk[0][2] - helping3*ijk[0][1], ijk[0][2], 0, helping1*ijk[0][2] - helping3*ijk[0][0], ijk[0][0], ijk[0][1], 0) / JGN_Det3x3(0, ijk[0][2], -ijk[0][1], ijk[0][2], 0, -ijk[0][0], ijk[0][0], ijk[0][1], ijk[0][2]);
		}

		}

		helping1 = CustomSurfaces[CustomSurfacesCount - 1][0];
		helping2 = CustomSurfaces[CustomSurfacesCount - 1][1];
		helping3 = CustomSurfaces[CustomSurfacesCount - 1][2];

		if (crystalk == 0)
		{
		if (JGN_Det3x3(ijk[1][0], ijk[1][1], ijk[1][2], ijk[1][2], 0, -ijk[1][0], ijk[1][1], -ijk[1][0], 0) != 0)
		{
		CustomSurfaces[CustomSurfacesCount - 1][0] = JGN_Det3x3(0, ijk[1][1], ijk[1][2], helping1*ijk[1][2] - helping3*ijk[1][0], 0, -ijk[1][0], helping1*ijk[1][1] - helping2*ijk[1][0], -ijk[1][0], 0) / JGN_Det3x3(ijk[1][0], ijk[1][1], ijk[1][2], ijk[1][2], 0, -ijk[1][0], ijk[1][1], -ijk[1][0], 0);
		CustomSurfaces[CustomSurfacesCount - 1][1] = JGN_Det3x3(ijk[1][0], 0, ijk[1][2], ijk[1][2], helping1*ijk[1][2] - helping3*ijk[1][0], -ijk[1][0], ijk[1][1], helping1*ijk[1][1] - helping2*ijk[1][0], 0) / JGN_Det3x3(ijk[1][0], ijk[1][1], ijk[1][2], ijk[1][2], 0, -ijk[1][0], ijk[1][1], -ijk[1][0], 0);
		CustomSurfaces[CustomSurfacesCount - 1][2] = JGN_Det3x3(ijk[1][0], ijk[1][1], 0, ijk[1][2], 0, helping1*ijk[1][2] - helping3*ijk[1][0], ijk[1][1], -ijk[1][0], helping1*ijk[1][1] - helping2*ijk[1][0]) / JGN_Det3x3(ijk[1][0], ijk[1][1], ijk[1][2], ijk[1][2], 0, -ijk[1][0], ijk[1][1], -ijk[1][0], 0);
		}
		else if (JGN_Det3x3(0, ijk[1][2], -ijk[1][1], ijk[1][0], ijk[1][1], ijk[1][2], ijk[1][1], -ijk[1][0], 0) != 0)
		{
		CustomSurfaces[CustomSurfacesCount - 1][0] = JGN_Det3x3(helping2*ijk[1][2] - helping3*ijk[1][1], ijk[1][2], -ijk[1][1], 0, ijk[1][1], ijk[1][2], helping1*ijk[1][1] - helping2*ijk[1][0], -ijk[1][0], 0) / JGN_Det3x3(0, ijk[1][2], -ijk[1][1], ijk[1][0], ijk[1][1], ijk[1][2], ijk[1][1], -ijk[1][0], 0);
		CustomSurfaces[CustomSurfacesCount - 1][1] = JGN_Det3x3(0, helping2*ijk[1][2] - helping3*ijk[1][1], -ijk[1][1], ijk[1][0], 0, ijk[1][2], ijk[1][1], helping1*ijk[1][1] - helping2*ijk[1][0], 0) / JGN_Det3x3(0, ijk[1][2], -ijk[1][1], ijk[1][0], ijk[1][1], ijk[1][2], ijk[1][1], -ijk[1][0], 0);
		CustomSurfaces[CustomSurfacesCount - 1][2] = JGN_Det3x3(0, ijk[1][2], helping2*ijk[1][2] - helping3*ijk[1][1], ijk[1][0], ijk[1][1], 0, ijk[1][1], -ijk[1][0], helping1*ijk[1][1] - helping2*ijk[1][0]) / JGN_Det3x3(0, ijk[1][2], -ijk[1][1], ijk[1][0], ijk[1][1], ijk[1][2], ijk[1][1], -ijk[1][0], 0);
		}
		else if (JGN_Det3x3(0, ijk[1][2], -ijk[1][1], ijk[1][2], 0, -ijk[1][0], ijk[1][0], ijk[1][1], ijk[1][2]) != 0)
		{
		CustomSurfaces[CustomSurfacesCount - 1][0] = JGN_Det3x3(helping2*ijk[1][2] - helping3*ijk[1][1], ijk[1][2], -ijk[1][1], helping1*ijk[1][2] - helping3*ijk[1][0], 0, -ijk[1][0], 0, ijk[1][1], ijk[1][2]) / JGN_Det3x3(0, ijk[1][2], -ijk[1][1], ijk[1][2], 0, -ijk[1][0], ijk[1][0], ijk[1][1], ijk[1][2]);
		CustomSurfaces[CustomSurfacesCount - 1][1] = JGN_Det3x3(0, helping2*ijk[1][2] - helping3*ijk[1][1], -ijk[1][1], ijk[1][2], helping1*ijk[1][2] - helping3*ijk[1][0], -ijk[1][0], ijk[1][0], 0, ijk[1][2]) / JGN_Det3x3(0, ijk[1][2], -ijk[1][1], ijk[1][2], 0, -ijk[1][0], ijk[1][0], ijk[1][1], ijk[1][2]);
		CustomSurfaces[CustomSurfacesCount - 1][2] = JGN_Det3x3(0, ijk[1][2], helping2*ijk[1][2] - helping3*ijk[1][1], ijk[1][2], 0, helping1*ijk[1][2] - helping3*ijk[1][0], ijk[1][0], ijk[1][1], 0) / JGN_Det3x3(0, ijk[1][2], -ijk[1][1], ijk[1][2], 0, -ijk[1][0], ijk[1][0], ijk[1][1], ijk[1][2]);
		}
		}

		helping1 = CustomSurfaces[CustomSurfacesCount - 1][0];
		helping2 = CustomSurfaces[CustomSurfacesCount - 1][1];
		helping3 = CustomSurfaces[CustomSurfacesCount - 1][2];

		if (crystall == 0)
		{
		if (JGN_Det3x3(ijk[2][0], ijk[2][1], ijk[2][2], ijk[2][2], 0, -ijk[2][0], ijk[2][1], -ijk[2][0], 0) != 0)
		{

		CustomSurfaces[CustomSurfacesCount - 1][0] = JGN_Det3x3(0, ijk[2][1], ijk[2][2], helping1*ijk[2][2] - helping3*ijk[2][0], 0, -ijk[2][0], helping1*ijk[2][1] - helping2*ijk[2][0], -ijk[2][0], 0) / JGN_Det3x3(ijk[2][0], ijk[2][1], ijk[2][2], ijk[2][2], 0, -ijk[2][0], ijk[2][1], -ijk[2][0], 0);
		CustomSurfaces[CustomSurfacesCount - 1][1] = JGN_Det3x3(ijk[2][0], 0, ijk[2][2], ijk[2][2], helping1*ijk[2][2] - helping3*ijk[2][0], -ijk[2][0], ijk[2][1], helping1*ijk[2][1] - helping2*ijk[2][0], 0) / JGN_Det3x3(ijk[2][0], ijk[2][1], ijk[2][2], ijk[2][2], 0, -ijk[2][0], ijk[2][1], -ijk[2][0], 0);
		CustomSurfaces[CustomSurfacesCount - 1][2] = JGN_Det3x3(ijk[2][0], ijk[2][1], 0, ijk[2][2], 0, helping1*ijk[2][2] - helping3*ijk[2][0], ijk[2][1], -ijk[2][0], helping1*ijk[2][1] - helping2*ijk[2][0]) / JGN_Det3x3(ijk[2][0], ijk[2][1], ijk[2][2], ijk[2][2], 0, -ijk[2][0], ijk[2][1], -ijk[2][0], 0);
		}
		else if (JGN_Det3x3(0, ijk[2][2], -ijk[2][1], ijk[2][0], ijk[2][1], ijk[2][2], ijk[2][1], -ijk[2][0], 0) != 0)
		{

		CustomSurfaces[CustomSurfacesCount - 1][0] = JGN_Det3x3(helping2*ijk[2][2] - helping3*ijk[2][1], ijk[2][2], -ijk[2][1], 0, ijk[2][1], ijk[2][2], helping1*ijk[2][1] - helping2*ijk[2][0], -ijk[2][0], 0) / JGN_Det3x3(0, ijk[2][2], -ijk[2][1], ijk[2][0], ijk[2][1], ijk[2][2], ijk[2][1], -ijk[2][0], 0);
		CustomSurfaces[CustomSurfacesCount - 1][1] = JGN_Det3x3(0, helping2*ijk[2][2] - helping3*ijk[2][1], -ijk[2][1], ijk[2][0], 0, ijk[2][2], ijk[2][1], helping1*ijk[2][1] - helping2*ijk[2][0], 0) / JGN_Det3x3(0, ijk[2][2], -ijk[2][1], ijk[2][0], ijk[2][1], ijk[2][2], ijk[2][1], -ijk[2][0], 0);
		CustomSurfaces[CustomSurfacesCount - 1][2] = JGN_Det3x3(0, ijk[2][2], helping2*ijk[2][2] - helping3*ijk[2][1], ijk[2][0], ijk[2][1], 0, ijk[2][1], -ijk[2][0], helping1*ijk[2][1] - helping2*ijk[2][0]) / JGN_Det3x3(0, ijk[2][2], -ijk[2][1], ijk[2][0], ijk[2][1], ijk[2][2], ijk[2][1], -ijk[2][0], 0);
		}
		else if (JGN_Det3x3(0, ijk[2][2], -ijk[2][1], ijk[2][2], 0, -ijk[2][0], ijk[2][0], ijk[2][1], ijk[2][2]) != 0)
		{

		CustomSurfaces[CustomSurfacesCount - 1][0] = JGN_Det3x3(helping2*ijk[2][2] - helping3*ijk[2][1], ijk[2][2], -ijk[2][1], helping1*ijk[2][2] - helping3*ijk[2][0], 0, -ijk[2][0], 0, ijk[2][1], ijk[2][2]) / JGN_Det3x3(0, ijk[2][2], -ijk[2][1], ijk[2][2], 0, -ijk[2][0], ijk[2][0], ijk[2][1], ijk[2][2]);
		CustomSurfaces[CustomSurfacesCount - 1][1] = JGN_Det3x3(0, helping2*ijk[2][2] - helping3*ijk[2][1], -ijk[2][1], ijk[2][2], helping1*ijk[2][2] - helping3*ijk[2][0], -ijk[2][0], ijk[2][0], 0, ijk[2][2]) / JGN_Det3x3(0, ijk[2][2], -ijk[2][1], ijk[2][2], 0, -ijk[2][0], ijk[2][0], ijk[2][1], ijk[2][2]);
		CustomSurfaces[CustomSurfacesCount - 1][2] = JGN_Det3x3(0, ijk[2][2], helping2*ijk[2][2] - helping3*ijk[2][1], ijk[2][2], 0, helping1*ijk[2][2] - helping3*ijk[2][0], ijk[2][0], ijk[2][1], 0) / JGN_Det3x3(0, ijk[2][2], -ijk[2][1], ijk[2][2], 0, -ijk[2][0], ijk[2][0], ijk[2][1], ijk[2][2]);
		}
		}*/


		helping1 = CustomSurfaces[CustomSurfacesCount - 1][0];
		helping2 = CustomSurfaces[CustomSurfacesCount - 1][1];
		helping3 = CustomSurfaces[CustomSurfacesCount - 1][2];

		CustomSurfaces[CustomSurfacesCount - 1][0] = CustomSurfaces[CustomSurfacesCount - 1][0] / sqrt(pow(helping1, 2) + pow(helping2, 2) + pow(helping3, 2));
		CustomSurfaces[CustomSurfacesCount - 1][1] = CustomSurfaces[CustomSurfacesCount - 1][1] / sqrt(pow(helping1, 2) + pow(helping2, 2) + pow(helping3, 2));
		CustomSurfaces[CustomSurfacesCount - 1][2] = CustomSurfaces[CustomSurfacesCount - 1][2] / sqrt(pow(helping1, 2) + pow(helping2, 2) + pow(helping3, 2));

		help += 2;
		isngtv = 0;



		if (help[0] >= 48 && help[0] <= 57)
		{
			CustomSurfaces[CustomSurfacesCount - 1][3] = help[0] - 48;
		}
		else if (help[0] == '-')
		{
			isngtv = 1;
		}
		else
		{
			okrender = 0;
			return;
		}
		help = help + 2;
		while (help[0] != ')' && loop < 6)

		{
			loop++;
			if (help[0] >= 48 && help[0] <= 57)
			{
				CustomSurfaces[CustomSurfacesCount - 1][3] = 10 * CustomSurfaces[CustomSurfacesCount - 1][3] + help[0] - 48;
				help += 2;
			}
			else
			{
				okrender = 0;
				return;
			}
		}
		loop = 0;
		if (isngtv)
		{
			CustomSurfaces[CustomSurfacesCount - 1][3] = -CustomSurfaces[CustomSurfacesCount - 1][3];
		}
		help += 2;
		isngtv = 0;


		if (okrender == 0)
		{
			CustomSurfacesCount--;
		}

		goto peintit;


	}



	//"PlaneH("
	/*for (i = 0; i < 7; i++)
	{
	if (test5[i] == ttt[i])
	{

	}
	else
	{
	i = 100;
	}
	}
	if (i == 7)
	{
	okrender = 1;
	CustomSurfacesCount++;

	CustomSurfaces = (float**)realloc(CustomSurfaces, sizeof(float*)*CustomSurfacesCount);
	CustomSurfaces[CustomSurfacesCount - 1] = (float*)malloc(sizeof(float) * 4);
	CustomSurfaces[CustomSurfacesCount - 1][0] = 0;
	CustomSurfaces[CustomSurfacesCount - 1][1] = 0;
	CustomSurfaces[CustomSurfacesCount - 1][2] = 0;
	CustomSurfaces[CustomSurfacesCount - 1][3] = 0;

	int u = 0;
	int v = 0;

	help = (char*)(ttt + 7);
	if (help[0] >= 48 && help[0] <= 57)
	{
	u = help[0] - 48;
	}
	else if (help[0] == '-')
	{
	isngtv = 1;
	}
	else
	{
	okrender = 0;
	}
	help = help + 2;
	loop = 0;
	while (help[0] != ',' && loop<6)
	{
	loop++;
	if (help[0] >= 48 && help[0] <= 57)
	{
	u = 10 * u + help[0] - 48;
	help += 2;
	}
	else
	{
	okrender = 0;


	}
	}
	loop = 0;
	if (isngtv)
	{
	u = -u;
	}
	help += 2;
	isngtv = 0;


	if (help[0] >= 48 && help[0] <= 57)
	{
	v = help[0] - 48;
	}
	else if (help[0] == '-')
	{
	isngtv = 1;
	}
	else
	{
	okrender = 0;

	}
	help = help + 2;
	while (help[0] != ',' && loop<6)

	{
	loop++;
	if (help[0] >= 48 && help[0] <= 57)
	{
	v = 10 * v + help[0] - 48;
	help += 2;
	}
	else
	{
	okrender = 0;


	}
	}
	loop = 0;
	if (isngtv)
	{
	v = -v;
	}
	help += 2;
	isngtv = 0;

	CustomSurfaces[CustomSurfacesCount - 1][0] = u;
	CustomSurfaces[CustomSurfacesCount - 1][1] = u/sqrt(3) + v*2.0 / sqrt(3);

	if (help[0] >= 48 && help[0] <= 57)
	{
	CustomSurfaces[CustomSurfacesCount - 1][2] = help[0] - 48;
	}
	else if (help[0] == '-')
	{
	isngtv = 1;
	}
	else
	{
	okrender = 0;

	}
	help = help + 2;
	while (help[0] != ',' && loop<6)

	{
	loop++;
	if (help[0] >= 48 && help[0] <= 57)
	{
	CustomSurfaces[CustomSurfacesCount - 1][2] = 10 * CustomSurfaces[CustomSurfacesCount - 1][2] + help[0] - 48;
	help += 2;
	}
	else
	{
	okrender = 0;

	}
	}
	loop = 0;
	if (isngtv)
	{
	CustomSurfaces[CustomSurfacesCount - 1][2] = -CustomSurfaces[CustomSurfacesCount - 1][2];
	}
	help += 2;
	isngtv = 0;



	if (help[0] >= 48 && help[0] <= 57)
	{
	CustomSurfaces[CustomSurfacesCount - 1][3] = help[0] - 48;
	}
	else if (help[0] == '-')
	{
	isngtv = 1;
	}
	else
	{
	okrender = 0;

	}
	help = help + 2;
	while (help[0] != ')' && loop<6)

	{
	loop++;
	if (help[0] >= 48 && help[0] <= 57)
	{
	CustomSurfaces[CustomSurfacesCount - 1][3] = 10 * CustomSurfaces[CustomSurfacesCount - 1][3] + help[0] - 48;
	help += 2;
	}
	else
	{
	okrender = 0;

	}
	}
	loop = 0;
	if (isngtv)
	{
	CustomSurfaces[CustomSurfacesCount - 1][3] = -CustomSurfaces[CustomSurfacesCount - 1][3];
	}
	help += 2;
	isngtv = 0;


	if (okrender == 0)
	{
	CustomSurfacesCount--;
	}
	JGN_PostRedisplay();

	goto peintit;

	}*/





	//char *test2 = "Clean";
	for (i = 0; i < 5; i++)
	{
		if (test1[1][i] == ttt[i])
		{

		}
		else
		{
			i = 100;
		}
	}

	if (i == 5)
	{
		okrender = 1;
		CustomSurfacesCount = 0;
		for (i = 0; i < CustomSurfacesCount; i++)
		{
			delete(CustomSurfaces[i]);
		}
		delete(CustomSurfaces);
		CustomSurfaces = NULL;
		goto peintit;



	}

	//char *test3 = "Undo";
	for (i = 0; i < 4; i++)
	{
		if (test1[2][i] == ttt[i])
		{

		}
		else
		{
			i = 100;
		}
	}
	if (i == 4)
	{
		okrender = 1;
		CustomSurfacesCount--;
		if (CustomSurfacesCount < 0)
		{
			CustomSurfacesCount = 0;
		}

		goto peintit;


	}
	//char *test4 = "spheres";
	for (i = 0; i < 7; i++)
	{
		if (test1[6][i] == ttt[i])
		{

		}
		else
		{
			i = 100;
		}
	}
	if (i == 7)
	{
		okrender = 1;
		shperes_on = 1;
		glEnable(GL_LIGHTING);

		sphStacks = 2;
		sphSides = 3;

		goto peintit;

	}

	//char *test4 = "points";
	for (i = 0; i < 6; i++)
	{
		if (test1[7][i] == ttt[i])
		{

		}
		else
		{
			i = 100;
		}
	}
	if (i == 6)
	{
		okrender = 1;
		shperes_on = 0;
		glDisable(GL_LIGHTING);

		goto peintit;

	}

	//char *test4 = "SuperCell(";
	for (i = 0; i < 10; i++)
	{
		if (test1[3][i] == ttt[i])
		{

		}
		else
		{
			i = 100;
		}
	}
	if (i == 10)
	{
		okrender = 1;
		help = (char*)(ttt + 10);
		if (help[0] >= 48 && help[0] <= 57)
		{
			sizedprotector[0] = help[0] - 48;
		}
		else
		{
			okrender = 0;
			return;
		}
		help = help + 2;
		while (help[0] != ',' && loop < 6)
		{
			loop++;
			if (help[0] >= 48 && help[0] <= 57)
			{
				sizedprotector[0] = 10 * sizedprotector[0] + help[0] - 48;
				help += 2;
			}
			else
			{
				okrender = 0;
				return;

			}
		}
		loop = 0;
		help += 2;




		if (help[0] >= 48 && help[0] <= 57)
		{
			sizedprotector[1] = help[0] - 48;
		}
		else
		{
			okrender = 0;
			return;
		}
		help = help + 2;
		while (help[0] != ',' && loop < 6)
		{
			loop++;
			if (help[0] >= 48 && help[0] <= 57)
			{
				sizedprotector[1] = 10 * sizedprotector[1] + help[0] - 48;
				help += 2;
			}
			else
			{
				okrender = 0;
				return;

			}
		}
		loop = 0;
		help += 2;



		if (help[0] >= 48 && help[0] <= 57)
		{
			sizedprotector[2] = help[0] - 48;
		}
		else
		{
			okrender = 0;
			return;
		}
		help = help + 2;
		while (help[0] != ')' && loop < 6)
		{
			loop++;
			if (help[0] >= 48 && help[0] <= 57)
			{
				sizedprotector[2] = 10 * sizedprotector[2] + help[0] - 48;
				help += 2;
			}
			else
			{
				okrender = 0;

				return;
			}
		}
		loop = 0;
		help += 2;



		if (okrender)
		{


			xexe[0] = 0;
			xexe[1] = 0;
			xexe[2] = 0;

			sized[0] = sizedprotector[0];
			sized[1] = sizedprotector[1];
			sized[2] = sizedprotector[2];
			custom_sized[0] = sizedprotector[0];
			custom_sized[1] = sizedprotector[1];
			custom_sized[2] = sizedprotector[2];

			jgn_supercell_xyz[0] = sized[0];
			jgn_supercell_xyz[1] = sized[1];
			jgn_supercell_xyz[2] = sized[2];

			if (sized[0] / 2 != sized[0] / 2.0)
			{
				xexe[0] = 1;
			}

			if (sized[1] / 2 != sized[1] / 2.0)
			{
				xexe[1] = 1;
			}

			if (sized[2] / 2 != sized[2] / 2.0)
			{
				xexe[2] = 1;
			}
			//periodic_table = fopen("periodic_table.jgn", "r");

			crystal = (float*)realloc(NULL, sizeof(float)*(jgn_supercell_xyz[0] * jgn_supercell_xyz[1] * jgn_supercell_xyz[2] * t * 5));
			crystal_backup = (float*)realloc(NULL, sizeof(float)*(jgn_supercell_xyz[0] * jgn_supercell_xyz[1] * jgn_supercell_xyz[2] * t * 5));
			selective_dynamics = (char*)realloc(selective_dynamics, sizeof(char)*(jgn_supercell_xyz[0] * jgn_supercell_xyz[1] * jgn_supercell_xyz[2] * t * 3));

			//	crystal_backup = (float*)realloc(NULL, sizeof(float)*(jgn_supercell_xyz[0] * jgn_supercell_xyz[1] * jgn_supercell_xyz[2] * t * 5));
			//double ttest = omp_get_wtime();

#pragma omp parallel for firstprivate(jgn_supercell_xyz, xexe, my_direct, inptype, uccartesian, ijk, t)
			for (int ole3 = 0; ole3 < t; ole3++) {//for every atom in the unit cell
				for (int ole4 = -jgn_supercell_xyz[2] / 2; ole4 < jgn_supercell_xyz[2] / 2 + xexe[2]; ole4++) {
					for (int ole2 = -jgn_supercell_xyz[1] / 2; ole2 < jgn_supercell_xyz[1] / 2 + xexe[1]; ole2++) {
						for (int ole = -jgn_supercell_xyz[0] / 2; ole < jgn_supercell_xyz[0] / 2 + xexe[0]; ole++) {

							crystal[0 + 5 * (ole3 + t * (ole4 + jgn_supercell_xyz[2] / 2 + jgn_supercell_xyz[2] * (ole2 + jgn_supercell_xyz[1] / 2 + jgn_supercell_xyz[1] * (ole + jgn_supercell_xyz[0] / 2))))] = my_direct[0 + 5 * ole3];//atomikos ari8mos
						//	crystal_backup[0 + 5 * (ole3 + t*(ole4 + jgn_supercell_xyz[2] / 2 + jgn_supercell_xyz[2] * (ole2 + jgn_supercell_xyz[1] / 2 + jgn_supercell_xyz[1] * (ole + jgn_supercell_xyz[0] / 2))))] = my_direct[0 + 5 * ole3];//atomikos ari8mos
						//selective dynamics
							selective_dynamics[0 + 3 * (ole3 + t * (ole4 + sized[2] / 2 + sized[2] * (ole2 + sized[1] / 2 + sized[1] * (ole + sized[0] / 2))))] = selective_dynamics[0 + 3 * (ole3)];
							selective_dynamics[1 + 3 * (ole3 + t * (ole4 + sized[2] / 2 + sized[2] * (ole2 + sized[1] / 2 + sized[1] * (ole + sized[0] / 2))))] = selective_dynamics[1 + 3 * (ole3)];
							selective_dynamics[2 + 3 * (ole3 + t * (ole4 + sized[2] / 2 + sized[2] * (ole2 + sized[1] / 2 + sized[1] * (ole + sized[0] / 2))))] = selective_dynamics[2 + 3 * (ole3)];

							crystal[1 + 5 * (ole3 + t * (ole4 + jgn_supercell_xyz[2] / 2 + jgn_supercell_xyz[2] * (ole2 + jgn_supercell_xyz[1] / 2 + jgn_supercell_xyz[1] * (ole + jgn_supercell_xyz[0] / 2))))] = my_direct[1 + 5 * ole3];//atomiko varos
						//	crystal_backup[1 + 5 * (ole3 + t*(ole4 + jgn_supercell_xyz[2] / 2 + jgn_supercell_xyz[2] * (ole2 + jgn_supercell_xyz[1] / 2 + jgn_supercell_xyz[1] * (ole + jgn_supercell_xyz[0] / 2))))] = my_direct[1 + 5 * ole3];//atomiko varos
							if (inptype == 'd')
							{
								//crystal[ole + sized[0] / 2][ole2 + sized[1] / 2][ole4 + sized[2] / 2][ole3][1] = direct[1+5*ole3];
								for (int ole1 = 0; ole1 < 3; ole1++) {
									crystal[ole1 + 2 + 5 * (ole3 + t * (ole4 + sized[2] / 2 + sized[2] * (ole2 + sized[1] / 2 + sized[1] * (ole + sized[0] / 2))))] = uccartesian[ole1 + 3 * ole3] + ole * ijk[0][ole1] + ole2 * ijk[1][ole1] + ole4 * ijk[2][ole1];//cartesians
									crystal_backup[ole1 + 2 + 5 * (ole3 + t * (ole4 + sized[2] / 2 + sized[2] * (ole2 + sized[1] / 2 + sized[1] * (ole + sized[0] / 2))))] = uccartesian[ole1 + 3 * ole3] + ole * ijk[0][ole1] + ole2 * ijk[1][ole1] + ole4 * ijk[2][ole1];//cartesians

									//if (crystal[ole1 + 2 + 5 * (ole3 + t * (ole4 + sized[2] / 2 + sized[2] * (ole2 + sized[1] / 2 + sized[1] * (ole + sized[0] / 2))))] < min_xyz[ole1])
									//{
									//	min_xyz[ole1] = crystal[ole1 + 2 + 5 * (ole3 + t * (ole4 + sized[2] / 2 + sized[2] * (ole2 + sized[1] / 2 + sized[1] * (ole + sized[0] / 2))))];

									//}
									//if (crystal[ole1 + 2 + 5 * (ole3 + t * (ole4 + sized[2] / 2 + sized[2] * (ole2 + sized[1] / 2 + sized[1] * (ole + sized[0] / 2))))] > max_xyz[ole1])
									//{
									//	max_xyz[ole1] = crystal[ole1 + 2 + 5 * (ole3 + t * (ole4 + sized[2] / 2 + sized[2] * (ole2 + sized[1] / 2 + sized[1] * (ole + sized[0] / 2))))];

									//}

								}

							}
							else if (inptype == 'c')
							{
								float oleh[3] = { ole,ole2,ole4 };
								for (ole1 = 0; ole1 < 3; ole1++) {
									if (oleh[ole1] >= 0)
										oleh[ole1]++;
									crystal[ole1 + 2 + 5 * (ole3 + t * (ole4 + jgn_supercell_xyz[2] / 2 + jgn_supercell_xyz[2] * (ole2 + jgn_supercell_xyz[1] / 2 + jgn_supercell_xyz[1] * (ole + jgn_supercell_xyz[0] / 2))))] = my_direct[ole1 + 2 + 5 * (ole3)] + ole * ijk[0][ole1] + ole2 * ijk[1][ole1] + ole4 * ijk[2][ole1];//cartesians
									crystal_backup[ole1 + 2 + 5 * (ole3 + t * (ole4 + jgn_supercell_xyz[2] / 2 + jgn_supercell_xyz[2] * (ole2 + jgn_supercell_xyz[1] / 2 + jgn_supercell_xyz[1] * (ole + jgn_supercell_xyz[0] / 2))))] = my_direct[ole1 + 2 + 5 * (ole3)] + ole * ijk[0][ole1] + ole2 * ijk[1][ole1] + ole4 * ijk[2][ole1];//cartesians

	/*								if (crystal[ole1 + 2 + 5 * (ole3 + t * (ole4 + sized[2] / 2 + sized[2] * (ole2 + sized[1] / 2 + sized[1] * (ole + sized[0] / 2))))] < min_xyz[ole1])
									{
										min_xyz[ole1] = crystal[ole1 + 2 + 5 * (ole3 + t * (ole4 + sized[2] / 2 + sized[2] * (ole2 + sized[1] / 2 + sized[1] * (ole + sized[0] / 2))))];

									}
									if (crystal[ole1 + 2 + 5 * (ole3 + t * (ole4 + sized[2] / 2 + sized[2] * (ole2 + sized[1] / 2 + sized[1] * (ole + sized[0] / 2))))] > max_xyz[ole1])
									{
										max_xyz[ole1] = crystal[ole1 + 2 + 5 * (ole3 + t * (ole4 + sized[2] / 2 + sized[2] * (ole2 + sized[1] / 2 + sized[1] * (ole + sized[0] / 2))))];

									}*/

								}
							}


							//rewind(periodic_table);

						}
					}
				}
			}
			//ttest =  omp_get_wtime() - ttest;
			//cout << ttest << endl;
			//fclose(periodic_table);


		}

		goto peintit;

	}


peintit:

	DestroyWindow(CommandTextField);



	CommandTextField = CreateWindow(L"EDIT",
		0, WS_VISIBLE | WS_CHILD | WS_BORDER | ES_MULTILINE | ES_AUTOHSCROLL,
		0, 735, 300, 40,
		hWndCommandLine, NULL, NULL, NULL);

	oldEditProc = (WNDPROC)SetWindowLongPtr(CommandTextField, GWLP_WNDPROC, (LONG_PTR)WndProcEditBox);

	SendMessage(CommandTextField, WM_SETFONT, (WPARAM)fOnt, TRUE);

peintit1:


	if (okrender)
	{
		SendMessage(CommandTextHistory, EM_SETREADONLY, FALSE, NULL);

		okrender = 0;

		SendMessage(CommandTextHistory, EM_SETSEL, 0, -1);
		SendMessage(CommandTextHistory, EM_SETSEL, -1, 0);

		for (i = 0; i < 50; i++)
		{

			if (i == 0 && ttt[0] == 13)
			{

			}
			else
			{
				if (ttt[i] == 0)
				{
					i = 100;
				}
				else
				{
					SendMessage(CommandTextHistory, WM_CHAR, (TCHAR)ttt[i], 0);


				}
			}
		}

		if (d == 1)
		{
			SendMessage(CommandTextHistory, WM_CHAR, (TCHAR)ucender[0], 0);
		}

		SetFocus(CommandTextField);
		SendMessage(CommandTextHistory, EM_SETREADONLY, TRUE, NULL);

		for (i = 0; i < 50; i++)
		{
			CommandBuffer[i] = 0;

		}

		JGN_PostRedisplay();

	}
	else
	{
		SetFocus(CommandTextField);
	}

}